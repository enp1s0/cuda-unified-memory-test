#include <iostream>
#include <chrono>
#include <hipblas.h>
#include <hipblas.h>

constexpr std::size_t N = 1lu << 15;

std::size_t gemm_test() {
	float *da, *db, *dc;
	hipMalloc(&da, sizeof(float) * N * N);
	hipMalloc(&db, sizeof(float) * N * N);
	hipMalloc(&dc, sizeof(float) * N * N);

	float *ha, *hb, *hc;
	hipHostMalloc(&ha, sizeof(float) * N * N);
	hipHostMalloc(&hb, sizeof(float) * N * N);
	hipHostMalloc(&hc, sizeof(float) * N * N);

	// Init
	for (std::size_t i = 0; i < N * N; i++) {
		ha[i] = 1.f;
		hb[i] = 1.f;
		hc[i] = 1.f;
	}

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	const auto start_clock = std::chrono::system_clock::now();
	const float alpha = 1.0f;
	const float beta = 1.0f;

	// Copy to device
	hipMemcpy(da, ha, sizeof(float) * N * N, hipMemcpyDefault);
	hipMemcpy(db, hb, sizeof(float) * N * N, hipMemcpyDefault);
	hipMemcpy(dc, hc, sizeof(float) * N * N, hipMemcpyDefault);

	// Gemm
	hipblasSgemm(
			cublas_handle,
			HIPBLAS_OP_T, HIPBLAS_OP_N,
			N, N, N,
			&alpha,
			da, N,
			db, N,
			&beta,
			dc, N
			);

	hipDeviceSynchronize();
	const auto end_clock = std::chrono::system_clock::now();

	hipHostFree(ha);
	hipHostFree(hb);
	hipHostFree(hc);

	hipFree(da);
	hipFree(db);
	hipFree(dc);

	return std::chrono::duration_cast<std::chrono::microseconds>(end_clock - start_clock).count();
}

std::size_t gemm_test_umem() {
	float *ua, *ub, *uc;
	hipMallocManaged(&ua, sizeof(float) * N * N);
	hipMallocManaged(&ub, sizeof(float) * N * N);
	hipMallocManaged(&uc, sizeof(float) * N * N);

	// Init
	for (std::size_t i = 0; i < N * N; i++) {
		ua[i] = 1.f;
		ub[i] = 1.f;
		uc[i] = 1.f;
	}

	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);

	const auto start_clock = std::chrono::system_clock::now();
	const float alpha = 1.0f;
	const float beta = 1.0f;

	// Gemm
	hipblasSgemm(
			cublas_handle,
			HIPBLAS_OP_T, HIPBLAS_OP_N,
			N, N, N,
			&alpha,
			ua, N,
			ub, N,
			&beta,
			uc, N
			);

	hipDeviceSynchronize();
	const auto end_clock = std::chrono::system_clock::now();

	hipHostFree(ua);
	hipHostFree(ub);
	hipHostFree(uc);

	return std::chrono::duration_cast<std::chrono::microseconds>(end_clock - start_clock).count();
}

int main() {
	std::printf("Matrix size : %lu x %lu\n", N, N);

	const auto gemm_mem_time = gemm_test();
	std::printf("[ mem] %luus\n", gemm_mem_time);

	const auto gemm_umem_time = gemm_test_umem();
	std::printf("[umem] %luus\n", gemm_umem_time);
}
